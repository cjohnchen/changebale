#include "hip/hip_runtime.h"
/*
    This file is part of Leela Zero.
    Copyright (C) 2017-2018 Gian-Carlo Pascutto and contributors

    Leela Zero is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    Leela Zero is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with Leela Zero.  If not, see <http://www.gnu.org/licenses/>.
*/

#include "config.h"

#include "Utils.h"
#include <sstream>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hipblas.h>

using namespace Utils;

#define OUT_KWG 8
#define OUT_BWG 8
#define OUTIN_KWG 2

#ifdef USE_OPENCL

int DivUp(int a, int b) { return (a + b - 1) / b; }

void CudaError(hipError_t status, const std::string& file, const std::string& line) {
  if (status != hipSuccess) {
    std::ostringstream stringStream;
    stringStream << "CUDA error: " << std::string(hipGetErrorString(status))
                 << " (" << file
                 << ": " << line << ")";
    throw std::runtime_error(stringStream.str());
  }
}


std::string CublasGetErrorString(hipblasStatus_t status) {
  switch (status) {
    case HIPBLAS_STATUS_SUCCESS:
      return "HIPBLAS_STATUS_SUCCESS";
    case HIPBLAS_STATUS_NOT_INITIALIZED:
      return "HIPBLAS_STATUS_NOT_INITIALIZED";
    case HIPBLAS_STATUS_ALLOC_FAILED:
      return "HIPBLAS_STATUS_ALLOC_FAILED";
    case HIPBLAS_STATUS_INVALID_VALUE:
      return "HIPBLAS_STATUS_INVALID_VALUE";
    case HIPBLAS_STATUS_ARCH_MISMATCH:
      return "HIPBLAS_STATUS_ARCH_MISMATCH";
    case HIPBLAS_STATUS_MAPPING_ERROR:
      return "HIPBLAS_STATUS_MAPPING_ERROR";
    case HIPBLAS_STATUS_EXECUTION_FAILED:
      return "HIPBLAS_STATUS_EXECUTION_FAILED";
    case HIPBLAS_STATUS_INTERNAL_ERROR:
      return "HIPBLAS_STATUS_INTERNAL_ERROR";
    case HIPBLAS_STATUS_NOT_SUPPORTED:
      return "HIPBLAS_STATUS_NOT_SUPPORTED";
    case HIPBLAS_STATUS_UNKNOWN:
      return "HIPBLAS_STATUS_UNKNOWN";
  }
  return "Unknown cuBlas error";
}


void CublasError(hipblasStatus_t status, const std::string& file, const std::string& line) {
  if (status != HIPBLAS_STATUS_SUCCESS) {
    std::ostringstream stringStream;
    stringStream << "Cublas error: " << CublasGetErrorString(status)
                 << " (" << file
                 << ": " << line << ")";
    throw std::runtime_error(stringStream.str());
  }
}

#define STRINGIFY(x) #x
#define TOSTRING(x) STRINGIFY(x)
#define ReportCUBLASErrors(status) CublasError((status), __FILE__, TOSTRING(__LINE__))
#define ReportCUDAErrors(status) CudaError((status), __FILE__, TOSTRING(__LINE__))

// Winograd defines. TODO: include from Network.h

// Winograd filter transformation changes 3x3 filters to M + 3 - 1
#define WINOGRAD_M 4
#define WINOGRAD_ALPHA (WINOGRAD_M + 3 - 1)
#define WTILES (BOARD_SIZE / WINOGRAD_M + (BOARD_SIZE % WINOGRAD_M != 0))
#define WINOGRAD_TILE (WINOGRAD_ALPHA * WINOGRAD_ALPHA)
#define WINOGRAD_P (WTILES * WTILES)

#include "kernels/opencl_to_cuda.h"
#include "kernels/common.opencl"
#include "kernels/convolve1.opencl"
#include "kernels/convolve3.opencl"

void in_transform_host(void* in, void* V,
                  const int C, const int Cpad,
                  const int Ppad, const int batch_size) {
    auto tiles = WINOGRAD_P;

    auto wgs = ceilMultiple(batch_size * tiles, 32);

    dim3 threads( ceilMultiple(tiles, 32), 1 );
    dim3 grid( DivUp(wgs, threads.x), DivUp(C, threads.y) );

    in_transform<<<grid, threads>>>((float*)in, (float*)V, C, Cpad, Ppad, batch_size);

    ReportCUDAErrors(hipGetLastError());
}

void out_transform_host(void* M, void* Y,
                  const int K, const int Kpad,
                  const int Ppad, const int batch_size, void** residual,
				  void* means, void* stddivs) {
    auto tiles = WINOGRAD_P;

    auto wgs = ceilMultiple(batch_size * tiles, 32);

    dim3 threads( OUT_KWG, OUT_BWG );
    dim3 grid( DivUp(K, threads.x), DivUp(wgs, threads.y) );

	void *residual_device = nullptr;
	if (residual != nullptr) {
		residual_device = *residual;
	}

    out_transform_fused_bn<<<grid, threads>>>((float*)M, (float*)Y, K, Kpad, Ppad, batch_size, (float*)residual_device, (float*)means, (float*)stddivs);

    ReportCUDAErrors(hipGetLastError());
}

void fused_out_in_transform_host(void* M, void* Y, void* V,
                                 const int K, const int Kpad,
								 const int Ppad, const int Cpad,
                                 void** residual, void* means, void* stddivs,
                                 const int batch_size) {
    auto tiles = WINOGRAD_P;

    auto wgs_single = ceilMultiple(tiles, 32);

    dim3 threads( OUTIN_KWG, wgs_single, 1 );
    dim3 grid( DivUp(K, threads.x), 1, batch_size );

	void *residual_device = nullptr;
	if (residual != nullptr) {
		residual_device = *residual;
	}

    out_transform_fused_bn_in<<<grid, threads>>>((float*)M, (float*)Y, (float*)V, K, Kpad, Ppad, Cpad, (float*)residual_device, (float*)means, (float*)stddivs);

    ReportCUDAErrors(hipGetLastError());
}

void convolve1_host(int channels, int outputs,
				    void* bufferInput,
				    void* bufferOutput,
				    void* bufferMerge,
				    void* weights,
				    int batch_size) {
    // The size of the board is defined at compile time
    const int width = BOARD_SIZE;
    const int boardsize = NUM_INTERSECTIONS;
    const int rowTiles = BOARD_SIZE;

    // Input channel grouping in multiples of 8
    const int channelGroup = 8;
    const int channelShift = 3;
    const int rowGroup = 1;
    size_t outputGroup = std::min(outputs, 32);

    // Copy the rows locally
    size_t stripSize = width * sizeof(float);

    int rowBuffer = std::min<int>(channelGroup, 7);
    size_t rowSize = channelGroup * outputGroup * rowBuffer * sizeof(float);

    dim3 threads( channelGroup, outputGroup, rowGroup );
    dim3 grid( DivUp(channels, threads.x), DivUp(outputs, threads.y), DivUp(batch_size * rowTiles, threads.z));

	size_t shared_mem = stripSize * channelGroup * rowGroup + rowSize;

	convolve1<<<grid, threads, shared_mem>>>((float*)bufferInput, (float*)bufferMerge,
			(float*)weights);

    dim3 threads_merge( std::min(8, outputs), BOARD_SIZE, 1);
    dim3 grid_merge( DivUp(outputs, threads_merge.x), DivUp(NUM_INTERSECTIONS, threads_merge.y), DivUp(batch_size, threads_merge.z));

	merge<<<grid_merge, threads_merge>>>((float*)bufferMerge,
			(float*)bufferOutput, channels >> channelShift);

    ReportCUDAErrors(hipGetLastError());
//
//    try {
//        m_convolve_kernel->setArg(0, bufferInput);
//        m_convolve_kernel->setArg(1, bufferMerge);
//        m_convolve_kernel->setArg(2, weights[0]);
//        m_convolve_kernel->setArg(3, cl::Local(stripSize * channelGroup * rowGroup));
//        m_convolve_kernel->setArg(4, cl::Local(rowSize));
//
//        queue.enqueueNDRangeKernel(
//            *m_convolve_kernel, cl::NullRange,
//            cl::NDRange(channels, outputs, batch_size * rowTiles),
//            cl::NDRange(channelGroup, outputGroup, rowGroup));
//    } catch (const cl::Error &e) {
//        std::cerr << "Error in convolve1: " << e.what() << ": "
//                  << e.err() << std::endl;
//        throw;
//    }
//
//    cl::Kernel & merge_kernel = opencl_context.m_merge_kernel;
//    assert(channels % (1 << channelShift) == 0);
//
//    try {
//        merge_kernel.setArg(0, bufferMerge);
//        merge_kernel.setArg(1, bufferOutput);
//        merge_kernel.setArg(2, channels >> channelShift);
//
//        queue.enqueueNDRangeKernel(
//            merge_kernel, cl::NullRange,
//            cl::NDRange(outputs, boardsize, batch_size),
//            cl::NDRange(std::min(8, outputs), BOARD_SIZE, 1));
//    } catch (const cl::Error &e) {
//        std::cerr << "Error in merge: " << e.what() << ": "
//                  << e.err() << std::endl;
//        throw;
//    }
}

#endif
